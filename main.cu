
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

int cpu(const int* buf, int len);
int gpu(const int* buf, int len);

#define BUF_SIZE 100

int main(int argc, char **argv) {
	clock_t start, end;

	int* buf = (int*)malloc(sizeof(int) * BUF_SIZE);
	for(int i = 0;i < BUF_SIZE;i++) buf[i] = i;

	int result;

	start = clock();
	result = cpu(buf, BUF_SIZE);
	end = clock();
	printf("%d ms (res : %d)\n", (end - start), result);

	start = clock();
	result = gpu(buf, BUF_SIZE);
	end = clock();
	printf("%d ms (res : %d)\n", (end - start), result);
}
