#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define SHARED_MEMORY_MAX 10000
__global__ void kernel(int *buf, int len) {
    __shared__ int sm;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(len <= tid) return;

    if(tid == 0) sm = 0;

    __syncthreads();

    sm += buf[tid];

    __syncthreads();

    if(tid == 0) buf[0] = sm;
}

#define NUM_BLOCKS 1024
#define NUM_THREADS 512

int gpu(const int* buf, int len) {
	printf("[GPU Implement]\n");
	
	int* gbuf = NULL;
	hipMalloc((void**)&gbuf, sizeof(int) * len);
	hipMemcpy(gbuf, buf, sizeof(int) * len, hipMemcpyHostToDevice);
	
	kernel<<<NUM_BLOCKS, NUM_THREADS>>>(gbuf, len);

	int* nbuf = (int*)malloc(sizeof(int) * len);
	
	hipMemcpy(nbuf, gbuf, sizeof(int) * len, hipMemcpyDeviceToHost);
	int result = nbuf[0];

	hipFree(gbuf);
	free(nbuf);
	
	hipDeviceReset();

	return result;
}
